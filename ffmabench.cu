#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <Windows.h>

#define CHECK_STATUS() do { if (status != hipSuccess) \
	{\
	printf("CUDA call failed with %d at line %d\n", status, __LINE__);\
	return status;\
	} } while(0)

__device__ float4 operator*(float4 lhs, float4 rhs)
{
	float4 result = { lhs.x * rhs.x, lhs.y * rhs.y, lhs.z * rhs.z, lhs.w * rhs.w };
	return result;
}

__device__ float4 operator+(float4 lhs, float4 rhs)
{
	float4 result = { lhs.x + rhs.x, lhs.y + rhs.y, lhs.z + rhs.z, lhs.w + rhs.w };
	return result;
}

__device__ float4 operator-(float4 lhs, float4 rhs)
{
	float4 result = { lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z, lhs.w - rhs.w };
	return result;
}

#define LOOP_LEN 1000

// Each FFMA is 2 float operations
// Times 4 lanes times LOOP_LEN iterations times 2
__global__ void bench_kernel(hipTextureObject_t tex, float4 *out)
{
	float4 c0 = { 1.0f, 2.0f, 3.0f, 4.0f };
	float4 c1 = { 1.0f, 2.1f, 3.2f, 4.3f };
	float4 v0 = tex2D<float4>(tex, threadIdx.x, threadIdx.y);
	float4 v1 = v0;
	float4 v2;
	for (int i = 0; i < LOOP_LEN; i++)
	{
		v1 = v1 * v0 + c1;
	}
	v2 = v1;
	for (int i = 0; i < LOOP_LEN; i++)
	{
		v2 = v2 * v0 - c0;
	}
	*out = v2;
}

void* generate_arr_data()
{
	float *buffer = (float *)malloc(4 * 4 * 1024);
	for (int i = 0; i < 1024; i++)
	{
		buffer[i * 4 + 0] = 1.0f;
		buffer[i * 4 + 1] = 2.0f;
		buffer[i * 4 + 2] = 3.01f;
		buffer[i * 4 + 3] = 4.01f;
	}
	return buffer;
}

static inline int64_t get_ticks()
{
	LARGE_INTEGER ticks;
	if (!QueryPerformanceCounter(&ticks))
	{
		return 0;
	}
	return ticks.QuadPart;
}

static inline int64_t get_ticks_per_second()
{
	LARGE_INTEGER ticks;
	if (!QueryPerformanceFrequency(&ticks))
	{
		return 0;
	}
	return ticks.QuadPart;
}

int main()
{
	hipError_t status;
	hipArray_t arr;
	void* arr_data;
	hipChannelFormatDesc format_desc;
	hipTextureObject_t tex;
	hipResourceDesc res_desc;
	hipTextureDesc tex_desc;
	hipResourceViewDesc view_desc;
	void* out;
	double ticks_per_second = (double)get_ticks_per_second();
	double begin, end;
	const unsigned int blockSz = 1024;
	const unsigned int gridSz = 1048576;
	const unsigned int flopUnit = 16 * LOOP_LEN;

	format_desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindUnsigned);
	status = hipMallocArray(&arr, &format_desc, 1024, 1);
	CHECK_STATUS();
	arr_data = generate_arr_data();
	status = hipMemcpyToArray(arr, 0, 0, arr_data, 4 * 4 * 1024, hipMemcpyDefault);
	CHECK_STATUS();
	free(arr_data);
	res_desc.resType = hipResourceTypeArray;
	res_desc.res.array.array = arr;
	memset(&tex_desc, 0, sizeof(tex_desc));
	tex_desc.addressMode[0] = hipAddressModeBorder;
	tex_desc.addressMode[1] = hipAddressModeBorder;
	tex_desc.addressMode[2] = hipAddressModeBorder;
	tex_desc.readMode = hipReadModeElementType;
	view_desc.format = hipResViewFormatUnsignedInt4;
	view_desc.width = 1024;
	view_desc.height = 1;
	view_desc.depth = 0;
	view_desc.firstMipmapLevel = 0;
	view_desc.lastMipmapLevel = 0;
	view_desc.firstLayer = 0;
	view_desc.lastLayer = 0;
	status = hipCreateTextureObject(&tex, &res_desc, &tex_desc, &view_desc);
	CHECK_STATUS();
	hipMalloc(&out, 0x1000);
	printf("Running %u ops/thread * %u threads/block * %u blocks\n", flopUnit, blockSz, gridSz);
	begin = (double)get_ticks();
	bench_kernel<<<gridSz, blockSz>>>(tex, (float4 *)out);
	end = (double)get_ticks();
	printf("Launch latency: %f s\n", (end - begin) / ticks_per_second);
	status = hipDeviceSynchronize();
	CHECK_STATUS();
	end = (double)get_ticks();
	double duration = (end - begin) / ticks_per_second;
	printf("Kernel duration: %f s\n", duration);
	double tflops = (double)blockSz * gridSz * flopUnit / 1e12 / duration;
	printf("Your GPU's TFLOPS is %f\n", tflops);
	hipFree(out);
	status = hipDestroyTextureObject(tex);
	CHECK_STATUS();
	status = hipFreeArray(arr);
	CHECK_STATUS();
	return 0;
}
